
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#define rnd( x ) (x * rand() / RAND_MAX)
#define INF     2e10f

#define _bitsperpixel 32
#define _planes 1
#define _compression 0

#define _xpixelpermeter 0x13B //0x130B //2835 , 72 DPI
#define _ypixelpermeter 0x13B//0x130B //2835 , 72 DPI
#define pixel 0xFF
#pragma pack(push,1)
typedef struct{
    uint8_t signature[2];
    uint32_t filesize;
    uint32_t reserved;
    uint32_t fileoffset_to_pixelarray;
} fileheader;
typedef struct{
    uint32_t dibheadersize;
    uint32_t width;
    uint32_t height;
    uint16_t planes;
    uint16_t bitsperpixel;
    uint32_t compression;
    uint32_t imagesize;
    uint32_t ypixelpermeter;
    uint32_t xpixelpermeter;
    uint32_t numcolorspallette;
    uint32_t mostimpcolor;
} bitmapinfoheader;
typedef struct {
    fileheader fileheader;
    bitmapinfoheader bitmapinfoheader;
} bitmap;
#pragma pack(pop)
void genBpm (int height, int width, float *pixelbuffer_f) {
    uint32_t pixelbytesize = height*width*_bitsperpixel/8;
    uint32_t  _filesize =pixelbytesize+sizeof(bitmap);
    FILE *fp = fopen("test.bmp","wb");
    bitmap *pbitmap  = (bitmap*)calloc(1,sizeof(bitmap));

    int buffer_size = height*width*4;
    uint8_t *pixelbuffer = (uint8_t*)malloc(buffer_size);

    for(int i = 0; i<buffer_size;i++)
    {
     pixelbuffer[i]= (uint8_t) pixelbuffer_f[i];
    }


    //strcpy(pbitmap->fileheader.signature,"BM");
    pbitmap->fileheader.signature[0] = 'B';
    pbitmap->fileheader.signature[1] = 'M';
    pbitmap->fileheader.filesize = _filesize;
    pbitmap->fileheader.fileoffset_to_pixelarray = sizeof(bitmap);
    pbitmap->bitmapinfoheader.dibheadersize =sizeof(bitmapinfoheader);
    pbitmap->bitmapinfoheader.width = width;
    pbitmap->bitmapinfoheader.height = height;
    pbitmap->bitmapinfoheader.planes = _planes;
    pbitmap->bitmapinfoheader.bitsperpixel = _bitsperpixel;
    pbitmap->bitmapinfoheader.compression = _compression;
    pbitmap->bitmapinfoheader.imagesize = pixelbytesize;
    pbitmap->bitmapinfoheader.ypixelpermeter = _ypixelpermeter ;
    pbitmap->bitmapinfoheader.xpixelpermeter = _xpixelpermeter ;
    pbitmap->bitmapinfoheader.numcolorspallette = 0;
    fwrite (pbitmap, 1, sizeof(bitmap),fp);
    //memset(pixelbuffer,pixel,pixelbytesize);
    fwrite(pixelbuffer,1,pixelbytesize,fp);
    fclose(fp);
    free(pbitmap);
    free(pixelbuffer);
}


struct Sphere {
    float   r,b,g;
    float   radius;
    float   x,y,z;
};

void loadSpheres(Sphere *vet, int size, int dim, int radius, int sum){
   
	for (int i=0;i<size;i++){
			Sphere sphere;
            sphere.r = rnd(1);
            sphere.b = rnd(1);
            sphere.g = rnd(1);
            sphere.radius = rnd(radius) + sum;
            sphere.x = rnd(dim) - trunc(dim / 2);
            sphere.y = rnd(dim) - trunc(dim / 2);
            sphere.z = rnd(256) - 128;

            vet[i] = sphere;
            
           
        }
}

#define SPHERES 20

__global__ void kernel(int dim, Sphere * s,  float *ptr ) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;
    float   ox = (x - dim/2);
    float   oy = (y - dim/2);

    float   r=0, g=0, b=0;
    float   maxz = -99999;
    for(int i=0; i<SPHERES; i++) {
        float   n;
        float   t = -99999;
        float dx = ox - s[i].x;
        float dy = oy - s[i].y;
        float dz;
        if (dx*dx + dy*dy < s[i].radius * s[i].radius) {
            dz = sqrtf( s[i].radius * s[i].radius - dx*dx - dy*dy );
            n = dz / sqrtf( s[i].radius * s[i].radius );
            t = dz + s[i].z;

        } else {
            t = -99999;
        }
        if (t > maxz) {
              float fscale = n;
              r = s[i].r * fscale;
              g = s[i].g * fscale;
              b = s[i].b * fscale;
              maxz = t;
        }

    }

    ptr[offset*4 + 0] = (r * 255);
    ptr[offset*4 + 1] = (g * 255);
    ptr[offset*4 + 2] = (b * 255);
    ptr[offset*4 + 3] = 255;
}


int main(int argc, char *argv[]){
    int dim = atoi(argv[1]);
    //int sph = atoi(argv[2]);
    
    float   *final_image;
    float   *dev_image;
    Sphere * s;

    final_image = (float*) malloc(dim * dim * sizeof(float)*4);
    Sphere *temp_s = (Sphere*)malloc( sizeof(Sphere) * SPHERES );
    
    loadSpheres(temp_s, SPHERES, dim, 160, 20);

    /*

    if(dim == 256) {
      temp_s[0] = { 0.5647144993438521, 0.17026276436658833, 0.2513199255348369, 17.309945982238226, -83.67052217169714, -119.68724631488998, 98.2803430280465 };
      temp_s[1]=  { 0.9091158787804804, 0.1487777336954863, 0.1783196508682516, 21.85598315378277, -4.082155827509382, -0.5976744895779262, 24.65309610278635 };
      temp_s[2]=  { 0.6624347666859951, 0.3954588457899716, 0.6516922513504441, 17.61146885586108, 14.65279091769159, -110.39790032654805, 4.207159642323063 };
      temp_s[3]=  { 0.413251136814478, 0.3630481887264626, 0.1980040894802698, 16.984618671224098, -2.0039674062318795, -100.77260658589435, -95.8896450697348 };
      temp_s[4]=  { 0.13864558854945525, 0.9300515762810144, 0.6028931546983245, 12.94213690603351, -104.46021912289804, 28.098513748588516, 0.8711203344828675 };
      temp_s[5]=  { 0.21469771416364025, 0.9337748344370861, 0.33420819727164525, 18.591723380230107, -28.418836024048588, 107.64000366222115, 58.74007385479294 };
      temp_s[6]=  { 0.576219977416303, 0.6904812768944365, 0.7726371044038209, 18.319498275704213, -114.95272682882168, 88.7097384563738, -65.42777794732505 };
      temp_s[7]=  { 0.9437543870357372, 0.3283181249427778, 0.8446913052766503, 6.454512161626026, 122.41389202551346, 47.942869350260935, 121.83574938200019 };
      temp_s[8]=  { 0.8970305490279855, 0.014038514358958708, 0.9583117160557878, 18.243202002014222, 15.262184514908284, 94.37397381511886, -126.56245612964263 };
      temp_s[9]=  { 0.4650105288857692, 0.21561326944792017, 0.8502761925107578, 24.533677175206762, -43.872432630390335, -119.06222724082156, 61.88860744041261 };
      temp_s[10]=  { 0.9226660969878231, 0.9497665334025086, 0.8874477370525223, 21.117435224463637, -57.17752616962187, 77.29532761619922, -92.29578539384136 };
      temp_s[11]=  { 0.03280739768669698, 0.7397076326792199, 0.9098178044984283, 15.11871700186163, 26.442213202307187, 16.871608630634483, -61.63078707235938 };
      temp_s[12]=  { 0.565660573137608, 0.3304849391155736, 0.31153294473097937, 21.61976989043855, 26.27814569536423, -40.46607867671743, -1.0898770104068092 };
      temp_s[13]=  { 0.14319284646137884, 0.2749107333597827, 0.16772972808008058, 24.909054841761527, 78.25629444257942, 10.676107058931251, 48.06006042664876 };
      temp_s[14]=  { 0.007263405255287332, 0.7207861568041017, 0.14539017914365063, 17.106692709128087, -84.42054506057924, -53.30240791039766, 114.59334086123235 };
      temp_s[15]=  { 0.391155735953856, 0.3933835871456038, 0.4371471297341838, 7.766808069093905, 123.26548051393169, 54.50556962797938, 72.99832148197882 };
      temp_s[16]=  { 0.9168065431684317, 0.9289834284493546, 0.5631885738700522, 11.508377330851161, -9.691702017273471, 59.45103305154575, -26.8797265541551 };
      temp_s[17]=  { 0.06183050019837031, 0.08331553086947234, 0.8713950010681478, 18.9005706961272, -13.230872524185912, 60.95107882930998, -63.826166570024725 };
      temp_s[18]=  { 0.2659993285927915, 0.3164159062471389, 0.46769615771965695, 15.00518814661092, -103.35081026642659, -63.951170384838406, 4.4024781029694395 };
      temp_s[19]=  { 0.5646229438154241, 0.6811426129947813, 00.023316141239661855, 14.228797265541552, 21.32486953337198, 62.71675771355328, -123.35142063661611 };
    }
    if(dim == 1024){
      temp_s[0] = { 0.5647144993438521, 0.17026276436658833, 0.2513199255348369, 69.2397839289529, -334.6820886867886, -478.7489852595599, 98.2803430280465};
      temp_s[1] = { 0.9091158787804804, 0.1487777336954863, 0.1783196508682516, 87.42393261513108, -16.32862331003753, -2.390697958311705, 24.65309610278635};
      temp_s[2] = { 0.6624347666859951, 0.3954588457899716, 0.6516922513504441, 70.44587542344432, 58.61116367076636, -441.5916013061922, 4.207159642323063};
      temp_s[3] = { 0.413251136814478, 0.3630481887264626, 0.1980040894802698, 67.93847468489639, -8.015869624927518, -403.0904263435774, -95.8896450697348};
      temp_s[4] = { 0.13864558854945525, 0.9300515762810144, 0.6028931546983245, 51.76854762413404, -417.84087649159216, 112.39405499435406, 0.8711203344828675};
      temp_s[5] = { 0.21469771416364025, 0.9337748344370861, 0.33420819727164525, 74.36689352092043, -113.67534409619435, 430.5600146488846, 58.74007385479294};
      temp_s[6] = { 0.576219977416303, 0.6904812768944365, 0.7726371044038209, 73.27799310281685, -459.8109073152867, 354.8389538254952, -65.42777794732505};
      temp_s[7] = { 0.9437543870357372, 0.3283181249427778, 0.8446913052766503, 25.818048646504103, 489.65556810205385, 191.77147740104374, 121.83574938200019};
      temp_s[8] = { 0.8970305490279855, 0.014038514358958708, 0.9583117160557878, 72.97280800805689, 61.04873805963314, 377.49589526047544, -126.56245612964263};
      temp_s[9] = { 0.4650105288857692, 0.21561326944792017, 0.8502761925107578, 98.13470870082705, -175.48973052156134, -476.2489089632862, 61.88860744041261};
      temp_s[10] = { 0.9226660969878231, 0.9497665334025086, 0.8874477370525223, 84.46974089785455, -228.71010467848748, 309.18131046479687, -92.29578539384136};
      temp_s[11] = { 0.03280739768669698, 0.7397076326792199, 0.9098178044984283, 60.47486800744652, 105.76885280922875, 67.48643452253793, -61.63078707235938};
      temp_s[12] = { 0.565660573137608, 0.3304849391155736, 0.31153294473097937, 86.4790795617542, 105.11258278145692, -161.86431470686972, -1.0898770104068092};
      temp_s[13] = { 0.14319284646137884, 0.2749107333597827, 0.16772972808008058, 99.63621936704611, 313.02517777031767, 42.704428235725004, 48.06006042664876};
      temp_s[14] = { 0.007263405255287332, 0.7207861568041017, 0.14539017914365063, 68.42677083651235, -337.682180242317, -213.20963164159065, 114.59334086123235};
      temp_s[15] = { 0.391155735953856, 0.3933835871456038, 0.4371471297341838, 31.06723227637562, 493.06192205572677, 218.02227851191753, 72.99832148197882};
      temp_s[16] = { 0.9168065431684317, 0.9289834284493546, 0.5631885738700522, 46.033509323404644, -38.766808069093884, 237.804132206183, -26.8797265541551};
      temp_s[17] = { 0.06183050019837031, 0.08331553086947234, 0.8713950010681478, 75.6022827845088, -52.92349009674365, 243.8043153172399, -63.826166570024725};
      temp_s[18] = { 0.2659993285927915, 0.3164159062471389, 0.46769615771965695, 60.02075258644368, -413.40324106570637, -255.80468153935362, 4.4024781029694395};
      temp_s[19] = { 0.5646229438154241, 0.6811426129947813, 0.023316141239661855, 56.915189062166206, 85.29947813348792, 250.8670308542131, -123.35142063661611};
    }
    if (dim == 2048 || dim == 3072 || dim ==4096 || dim == 5120 || dim == 6144 || dim == 7168){
temp_s[0]=  {0.5647144993438521	,0.17026276436658833	 ,0.2513199255348369	 , 93.85967589342937	 , -669.3641773735771	 , -957.4979705191198	 , 98.2803430280465 };
temp_s[1]=	 { 0.9091158787804804	 ,0.1487777336954863	 ,0.1783196508682516	 , 121.13589892269661	 , -32.65724662007506	 , -4.78139591662341	 , 24.65309610278635};
temp_s[2]=	 {0.6624347666859951	 ,0.3954588457899716	 ,0.6516922513504441	 , 95.66881313516647	 , 117.22232734153272	 , -883.1832026123844	 , 4.207159642323063};
temp_s[3]=	 { 0.413251136814478	 ,0.3630481887264626	 , 0.1980040894802698	 , 91.90771202734459	 , -16.031739249855036	 , -806.1808526871548	 , -95.8896450697348};
temp_s[4]=	 { 0.13864558854945525	 , 0.9300515762810144	 , 0.6028931546983245	 , 67.65282143620107	 , -835.6817529831843	 , 224.78810998870813	 , 0.8711203344828675};
temp_s[5]=	 { 0.21469771416364025	 , 0.9337748344370861	 , 0.33420819727164525	 , 101.55034028138066	 , -227.3506881923887	 , 861.1200292977692	 , 58.74007385479294};
temp_s[6]=	 { 0.576219977416303	 , 0.6904812768944365	 , 0.7726371044038209	 , 99.91698965422529	 , -919.6218146305735	 , 709.6779076509904	 , -65.42777794732505};
temp_s[7]=	 { 0.9437543870357372	 , 0.3283181249427778	 , 0.8446913052766503	 , 28.727072969756158	 , 979.3111362041077	 , 383.5429548020875	 , 121.83574938200019};
temp_s[8]=	 { 0.8970305490279855	 , 0.014038514358958708	 , 0.9583117160557878	 , 99.45921201208533	 , 122.09747611926628	 , 754.9917905209509	 , -126.56245612964263};
temp_s[9]=	 { 0.4650105288857692	 , 0.21561326944792017	 , 0.8502761925107578	 , 137.20206305124057	 , -350.9794610431227	 , -952.4978179265725	 , 61.88860744041261};
temp_s[10]=	 { 0.9226660969878231	 , 0.9497665334025086	 , 0.8874477370525223	 , 116.70461134678182	 , -457.42020935697496	 , 618.3626209295937	 , -92.29578539384136};
temp_s[11]=	 { 0.03280739768669698	 , 0.7397076326792199	 , 0.9098178044984283	 , 80.71230201116978	 , 211.5377056184575	 , 134.97286904507587	 , -61.63078707235938};
temp_s[12]=	 { 0.565660573137608	 , 0.3304849391155736	 , 0.31153294473097937	 , 119.7186193426313	 , 210.22516556291384	 , -323.72862941373944	 , -1.0898770104068092};
temp_s[13]=	 { 0.14319284646137884	 , 0.2749107333597827	 , 0.16772972808008058	 , 139.45432905056919	 , 626.0503555406353	 , 85.40885647145001	 , 48.06006042664876};
temp_s[14]=	 { 0.007263405255287332	 ,0.7207861568041017	 , 0.14539017914365063	 , 92.64015625476851	 , -675.364360484634	 , -426.4192632831813	 , 114.59334086123235};
temp_s[15]=	 { 0.391155735953856	 , 0.3933835871456038	 , 0.4371471297341838	 , 36.60084841456343	 , 986.1238441114535	 , 436.04455702383507	 , 72.99832148197882};
temp_s[16]=	 { 0.9168065431684317	 , 0.9289834284493546	 , 0.5631885738700522	 , 59.050263985106966	 , -77.53361613818777	 , 475.608264412366	 , -26.8797265541551};
temp_s[17]=	 { 0.06183050019837031	 , 0.08331553086947234	 , 0.8713950010681478	 , 103.40342417676321	 , -105.8469801934873	 , 487.6086306344798	 , -63.826166570024725};
temp_s[18]=	 { 0.2659993285927915	 , 0.3164159062471389	 , 0.46769615771965695	 , 80.03112887966552	 , -826.8064821314127	 , -511.60936307870725	 , 4.4024781029694395};
temp_s[19]=	 { 0.5646229438154241	 ,0.6811426129947813	 , 0.023316141239661855	 , 75.3727835932493	 , 170.59895626697585	 , 501.7340617084262	 , -123.35142063661611};
    
        }
  
*/

    float time;
    hipEvent_t start, stop;   
    hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;

    hipMalloc( (void**)&dev_image, dim * dim * sizeof(float)*4);
    hipMalloc( (void**)&s, sizeof(Sphere) * SPHERES );
    
    hipMemcpy( s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice );

    dim3    grids(dim/16,dim/16);
    dim3    threads(16,16);

    kernel<<<grids,threads>>>(dim, s, dev_image);

    hipMemcpy( final_image, dev_image, dim * dim * sizeof(float) * 4,hipMemcpyDeviceToHost );
        
    hipFree( dev_image);
    hipFree( s );
    
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;

     printf("CUDA\t%d\t%3.1f\n", dim,time);
     genBpm(dim,dim,final_image);
    /*
    int height = dim;
    int width = dim;
    
    unsigned char* image = (unsigned char*) malloc(dim * dim *4); //[height][width][BYTES_PER_PIXEL];

    //double elapsed_time = ((double)(end_time - start_time) * 1000000.0) / CLOCKS_PER_SEC;

    char imageFileName[50];

    sprintf(imageFileName, "img-c-CUDAraytracer-%dx%d.bmp", dim, dim);
  

    int i, j;
    for (i = 0; i < height; i++) {
        for (j = 0; j < width; j++) {
            image[(i * dim + j) * 4 + 3] = final_image[(i * dim + j) * 4 + 3] ;
            image[(i * dim + j) * 4 + 0] = final_image[(i * dim + j) * 4 + 2] ;
            image[(i * dim + j) * 4 + 1] = final_image[(i * dim + j) * 4 + 1] ;
            image[(i * dim + j) * 4 + 2] = final_image[(i * dim + j) * 4 + 0] ;
        }
    }

    generateBitmapImage((unsigned char*) image, height, width, imageFileName);
    //printf("Image generated!!");

    //generateLog(elapsed_time, dim, sph, iteration);

    free(image);
    free(temp_s);
    free(final_image);
*/

}