#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include <string.h>
#include <math.h>
#define MAXBLOCKSIZE 512
#define BLOCK_SIZE_XY 4
int Size;
float *a, *b, *finalVec;
float *m;
// create both matrix and right hand side, Ke Wang 2013/08/12 11:51:06
void
create_matrix(float *m, int size){
  int i,j;
  float lamda = -0.01;
  float coe[2*size-1];
  float coe_i =0.0;
  for (i=0; i < size; i++)
    {
      coe_i = 10*exp(lamda*i); 
      j=size-1+i;     
      coe[j]=coe_i;
      j=size-1-i;     
      coe[j]=coe_i;
    }
  for (i=0; i < size; i++) {
      for (j=0; j < size; j++) {
	m[i*size+j]=coe[size-1-i+j];
      }
  }
}
int main(int argc, char *argv[])
{
          Size = atoi(argv[1]);
	      a = (float *) malloc(Size * Size * sizeof(float));
	      create_matrix(a, Size);
	      b = (float *) malloc(Size * sizeof(float));
	      for (int j =0; j< Size; j++)
	    	b[j]=1.0;
	      m = (float *) malloc(Size * Size * sizeof(float));
    InitPerRun();
	float time;
    hipEvent_t start, stop;   
     hipEventCreate(&start) ;
    hipEventCreate(&stop) ;
    hipEventRecord(start, 0) ;
    ForwardSub();
    hipEventRecord(stop, 0) ;
    hipEventSynchronize(stop) ;
    hipEventElapsedTime(&time, start, stop) ;
    printf("CUDA\t%d\t%3.1f\n", Size,time);
    BackSub();
   free(m);
    free(a);
    free(b);
}
void InitPerRun() 
{
	int i;
	for (i=0; i<Size*Size; i++)
			*(m+i) = 0.0;
}
__global__ void Fan1(float *m_cuda, float *a_cuda, int Size, int t)
{   
	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	*(m_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) = *(a_cuda+Size*(blockDim.x*blockIdx.x+threadIdx.x+t+1)+t) / *(a_cuda+Size*t+t);
}
__global__ void Fan2(float *m_cuda, float *a_cuda, float *b_cuda,int Size, int j1, int t)
{
	if(threadIdx.x + blockIdx.x * blockDim.x >= Size-1-t) return;
	if(threadIdx.y + blockIdx.y * blockDim.y >= Size-t) return;
    int xidx = blockIdx.x * blockDim.x + threadIdx.x;
	int yidx = blockIdx.y * blockDim.y + threadIdx.y;
		a_cuda[Size*(xidx+1+t)+(yidx+t)] -= m_cuda[Size*(xidx+1+t)+t] * a_cuda[Size*t+(yidx+t)];
	if(yidx == 0){
		b_cuda[xidx+1+t] -= m_cuda[Size*(xidx+1+t)+(yidx+t)] * b_cuda[t];
	}
}
void ForwardSub()
{
	int t;
    float *m_cuda,*a_cuda,*b_cuda;
	hipMalloc((void **) &m_cuda, Size * Size * sizeof(float));
	hipMalloc((void **) &a_cuda, Size * Size * sizeof(float));
	hipMalloc((void **) &b_cuda, Size * sizeof(float));	
	hipMemcpy(m_cuda, m, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(a_cuda, a, Size * Size * sizeof(float),hipMemcpyHostToDevice );
	hipMemcpy(b_cuda, b, Size * sizeof(float),hipMemcpyHostToDevice );
	int block_size,grid_size;
	block_size = MAXBLOCKSIZE;
	grid_size = (Size/block_size) + (!(Size%block_size)? 0:1);
	dim3 dimBlock(block_size);
	dim3 dimGrid(grid_size);
	int blockSize2d, gridSize2d;
	blockSize2d = BLOCK_SIZE_XY;
	gridSize2d = (Size/blockSize2d) + (!(Size%blockSize2d?0:1)); 
	dim3 dimBlockXY(blockSize2d,blockSize2d);
	dim3 dimGridXY(gridSize2d,gridSize2d);
	for (t=0; t<(Size-1); t++) {
		Fan1<<<dimGrid,dimBlock>>>(m_cuda,a_cuda,Size,t);
		hipDeviceSynchronize();
		Fan2<<<dimGridXY,dimBlockXY>>>(m_cuda,a_cuda,b_cuda,Size,Size-t,t);
		hipDeviceSynchronize();
		checkCUDAError("Fan2");
	}
	hipMemcpy(m, m_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(a, a_cuda, Size * Size * sizeof(float),hipMemcpyDeviceToHost );
	hipMemcpy(b, b_cuda, Size * sizeof(float),hipMemcpyDeviceToHost );
	hipFree(m_cuda);
	hipFree(a_cuda);
	hipFree(b_cuda);
}
void BackSub()
{
	finalVec = (float *) malloc(Size * sizeof(float));
	int i,j;
	for(i=0;i<Size;i++){
		finalVec[Size-i-1]=b[Size-i-1];
		for(j=0;j<i;j++)
		{
			finalVec[Size-i-1]-=*(a+Size*(Size-i-1)+(Size-j-1)) * finalVec[Size-j-1];
		}
		finalVec[Size-i-1]=finalVec[Size-i-1]/ *(a+Size*(Size-i-1)+(Size-i-1));
	}
}